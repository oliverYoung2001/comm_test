#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include <assert.h>
#include <hipblas.h>
#include <cstdint>
#include <hiprand.h>
#include <mpi.h>
#include <string>
#include <vector>
#include <time.h>
#include <sys/time.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include "comm.h"
#include <assert.h>
#include <cmath>
#include <chrono>
#include <unistd.h>
#include <iostream>
#include "json/json.h"
#include <fstream>
#include "assert.h" 
// #include <format>    // need c++20
typedef long long LL;
int comm_size, rank;
int N_GPUs, GPU_VISIBLE;

PROC_PARAMS* pp;

// #define CHECK_RESULT
// #define PRINT_JSON
// #define RECORD_TABLE
// #define ENABLE_GPU_P2P       // 性能不一定好！！！ 单个P2P更好，但多个P2P不一定好

int TIMES = 1;
int WARMUP = 0;
const int MAGIC_FACTOR = pow(2, 5) * pow(3, 3) * pow(5, 2) * 7;     // 151200, for tests on different number of GPUs
// 62792 B

const int SIZEIDX_START = 6;
const int SIZEIDX_END = 7;

const int SIZES_LEN = 8;
const LL SIZES[SIZES_LEN] = {   // int = 4B
    1LL * 256,                  // 1KB      // 打不满带宽
    1LL * 1024 * 1,             // 4KB      // 打不满带宽
    1LL * 1024 * 2,             // 8KB     // 会高一些!!! (仅在某些情况下)
    1LL * 1024 * 4,             // 16KB     // 会高一些!!!  （最好）
    // 1LL * 1024 * 8,             // 32KB     // 会高一些!!!  （最好）
    1LL * 1024 * 16,            // 64KB     // 会高一些!!!  （最好）
    // 1LL * 1024 * 64,            // 256KB    // 趋于稳定
    1LL * 1024 * 256,           // 1MB
    // 1LL * 1024 * 1024 * 1,      // 4MB      // 打不满带宽
    1LL * 1024 * 1024 * 32,     // 128MB
    // 1LL * 1024 * 1024 * 64,     // 256MB
    1LL * 1024 * 1024 * 128,    // 512MB
    // 1LL * 1024 * 1024 * 256,    // 1GB
    // 1LL * 1024 * 1024 * 512,    // 2GB
    // 1LL * 1024 * 1024 * 1024,   // 4GB      // 用hipMemcpy，竟然有性能下降！！！
    // 1LL * 1024 * 1024 * 2048,   // 8GB
    // 1LL * 1024 * 1024 * 4096,   // 16GB
    // 1LL * 1024 * 1024 * 8192,   // OOM
};

// const int SIZES_LEN = 26;
// const LL SIZES[SIZES_LEN] = {   // int = 4B
//     1LL * 256,                  // 1KB
//     1LL * 512,                  // 2KB
//     1LL * 1024 * 1,             // 4KB
//     1LL * 1024 * 2,             // 8KB
//     1LL * 1024 * 4,             // 16KB
//     1LL * 1024 * 8,             // 32KB
//     1LL * 1024 * 16,            // 64KB
//     1LL * 1024 * 32,            // 128KB
//     1LL * 1024 * 64,            // 256KB
//     1LL * 1024 * 128,           // 512KB
//     1LL * 1024 * 256,           // 1MB
//     1LL * 1024 * 512,           // 2MB
//     1LL * 1024 * 1024 * 1,      // 4MB
//     1LL * 1024 * 1024 * 2,      // 8MB
//     1LL * 1024 * 1024 * 4,      // 16MB
//     1LL * 1024 * 1024 * 8,      // 32MB
//     1LL * 1024 * 1024 * 16,     // 64MB
//     1LL * 1024 * 1024 * 32,     // 128MB
//     1LL * 1024 * 1024 * 64,     // 256MB
//     1LL * 1024 * 1024 * 128,    // 512MB
//     1LL * 1024 * 1024 * 256,    // 1GB
//     1LL * 1024 * 1024 * 512,    // 2GB
//     1LL * 1024 * 1024 * 1024,   // 4GB
//     1LL * 1024 * 1024 * 2048,   // 8GB
//     1LL * 1024 * 1024 * 4096,   // 16GB
//     1LL * 1024 * 1024 * 8192,   // OOM
// };

// const int SIZES_LEN = 18;
// const LL SIZES[SIZES_LEN] = {           // int = 4B
//     (LL)MAGIC_FACTOR * 1,               // 590.6KB
//     (LL)MAGIC_FACTOR * 2,               
//     (LL)MAGIC_FACTOR * 4,           
//     (LL)MAGIC_FACTOR * 8,            
//     (LL)MAGIC_FACTOR * 16,             
//     (LL)MAGIC_FACTOR * 32,
//     (LL)MAGIC_FACTOR * 64,
//     (LL)MAGIC_FACTOR * 128,
//     (LL)MAGIC_FACTOR * 256,
//     (LL)MAGIC_FACTOR * 512,
//     (LL)MAGIC_FACTOR * 1024,
//     (LL)MAGIC_FACTOR * 1024 * 2,
//     (LL)MAGIC_FACTOR * 1024 * 4,
//     (LL)MAGIC_FACTOR * 1024 * 8,
//     (LL)MAGIC_FACTOR * 1024 * 16,
//     (LL)MAGIC_FACTOR * 1024 * 32,       // 18.46GB
//     (LL)MAGIC_FACTOR * 1024 * 64,       // 36.91GB
//     (LL)MAGIC_FACTOR * 1024 * 128,      // 73.82GB
// };

bool check_pattern(Json::Value pattern, int N_GPUs) {
    for (int k = 0; k < pattern.size(); ++ k) {
        if (std::max(pattern[k][0].asInt(), pattern[k][1].asInt()) >= N_GPUs) {
            return false;
        }
    }
    return true;
}

void devicesSyncAll(int N_GPUs) {
    for (int gpuid = 0; gpuid < N_GPUs; ++ gpuid) {
        CUDA_CHECK(hipSetDevice(gpuid));
        CUDA_CHECK(hipDeviceSynchronize());
    }
}

void barrier(std::string& BACKEND, int N_GPUs) {
    // if (BACKEND.compare("hipMemcpy") == 0) {
    //     for (int gpuid = 0; gpuid < N_GPUs; ++ gpuid) {
    //         CUDA_CHECK(hipSetDevice(gpuid));
    //         CUDA_CHECK(hipDeviceSynchronize());
    //     }
    // }
    if (BACKEND.compare("NCCL") == 0 || BACKEND.compare("MPI") == 0) {
        CUDA_CHECK(hipSetDevice(rank));
        CUDA_CHECK(hipDeviceSynchronize());
        MPI_Barrier(MPI_COMM_WORLD);
    }
    
}

// cudaMemcpy_comm
void cudaMemcpy_comm(Json::Value& pairs, int** send_buf, int** recv_buf, LL SIZE, \
               hipStream_t* streams, int rank, ncclComm_t comm, MPI_Request* mpi_request) {
    for (int k = 0; k < pairs.size(); ++ k) {
        // CUDA_CHECK(hipMemcpyAsync(recv_buf[pairs[k][1].asInt()], send_buf[pairs[k][0].asInt()], 
        //                             SIZE * sizeof(int), hipMemcpyDeviceToDevice, streams[k]));
        CUDA_CHECK(hipMemcpyPeerAsync(recv_buf[pairs[k][1].asInt()], pairs[k][1].asInt(), \
                                   send_buf[pairs[k][0].asInt()], pairs[k][0].asInt(), \
                                   SIZE * sizeof(int), streams[k]));                                // 两者性能相似
    }
}

// NCCL_comm
void NCCL_comm(Json::Value& pairs, int** send_buf, int** recv_buf, LL SIZE, \
               hipStream_t* streams, int rank, ncclComm_t comm, MPI_Request* mpi_request) {
    // printf("rank: %d", rank);
    // Json::FastWriter sw;
    // std::cout << sw.write(pairs);
    NCCL_CHECK(ncclGroupStart());
    for (int k = 0; k < pairs.size(); ++ k) {
        if (rank == pairs[k][0].asInt()) {
            NCCL_CHECK(ncclSend(send_buf[rank], SIZE, ncclInt32, pairs[k][1].asInt(), comm, streams[0]));
        }
        if (rank == pairs[k][1].asInt()) {
            NCCL_CHECK(ncclRecv(recv_buf[rank], SIZE, ncclInt32, pairs[k][0].asInt(), comm, streams[0]));
        }
    }
    NCCL_CHECK(ncclGroupEnd());
}

// MPI_comm
void MPI_comm(Json::Value& pairs, int** send_buf, int** recv_buf, LL SIZE, \
               hipStream_t* streams, int rank, ncclComm_t comm, MPI_Request* mpi_request) {
    int req_num = 0;
    for (int k = 0; k < pairs.size(); ++ k) {
        if (rank == pairs[k][0].asInt()) {
            MPI_Isend(send_buf[rank], SIZE, MPI_INT, pairs[k][1].asInt(), 0/*tag*/, MPI_COMM_WORLD, mpi_request + (req_num ++));
        }
        if (rank == pairs[k][1].asInt()) {
            MPI_Irecv(recv_buf[rank], SIZE, MPI_INT, pairs[k][0].asInt(), 0/*tag*/, MPI_COMM_WORLD, mpi_request + (req_num ++));
        }
    }
    // MPI_Wait(mpi_request, NULL);
    MPI_Waitall(req_num, mpi_request , nullptr);

}


void mixed_p2p_comm(Json::Value& pairs, int p2p_id, int** send_buf, int** recv_buf, LL SIZE, \
               hipStream_t* streams, int rank, ncclComm_t comm, MPI_Request* mpi_request) {
    // printf("rank: %d", rank);
    // Json::FastWriter sw;
    // std::cout << sw.write(pairs);

    CUDA_CHECK(hipSetDevice(rank));    // just for NCCL
    NCCL_CHECK(ncclGroupStart());
    for (int k = 0; k < pairs.size(); ++ k) {
        if (p2p_id / (int)pow(3, k) % 3 == 0) {     // NCCL
            if (rank == pairs[k][0].asInt()) {
                NCCL_CHECK(ncclSend(send_buf[rank], SIZE, ncclInt32, pairs[k][1].asInt(), comm, streams[0]));
            }
            if (rank == pairs[k][1].asInt()) {
                NCCL_CHECK(ncclRecv(recv_buf[rank], SIZE, ncclInt32, pairs[k][0].asInt(), comm, streams[0]));
            }
        } else {                                    // DMA
            if (rank != 0) {
                continue;
            }
            CUDA_CHECK(hipMemcpyPeerAsync(recv_buf[pairs[k][1].asInt()], pairs[k][1].asInt(), \
                                   send_buf[pairs[k][0].asInt()], pairs[k][0].asInt(), \
                                   SIZE * sizeof(int), streams[k]));
        }
    }
    NCCL_CHECK(ncclGroupEnd());
}

int main(int argc, char** argv) {
    if (argc < 4) {
        printf("Need at least 2 args: \"<command> <gpus> <backend> <cp_file>\"\n");
        return - 1;
    }
    //Get number of gpus in the node
    CUDA_CHECK(hipGetDeviceCount(&GPU_VISIBLE));
    N_GPUs = std::stoi(argv[1]);
    assert(N_GPUs <= GPU_VISIBLE);
    std::string BACKEND = argv[2];
    std::string cp_file = argv[3];

    pp = new PROC_PARAMS(N_GPUs);
    pp->BACKEND = BACKEND;

    setup_env(pp);

    // Read patterns
    Json::Reader reader;
	Json::Value root;
    // std::string cp_file = "csrc/configs/conflict_patterns.json";
    std::ifstream in(cp_file.c_str(), std::ios::binary);
    if (! in.is_open()) {
		std::cout << "Error OPENING FILE\n";
		return - 1;
	}
    if (! reader.parse(in, root)) {
        std::cout << "Error READING FILE\n";
		return - 2;
    }

    // Init cudaStream
    int max_pair_num = 0;
    for (int cp = 0; cp < root.size(); ++ cp) {
        max_pair_num = std::max(max_pair_num, (int)root[cp].size());
    }
    int STREAM_NUM = std::max(N_GPUs, max_pair_num) + N_GPUs;       // for both DMA and NCCL
    hipStream_t* streams = new hipStream_t[STREAM_NUM];
    for (int i = 0; i < STREAM_NUM; ++ i) {
        hipStreamCreate(&streams[i]);
    }

    // Init MPI_Request
    MPI_Request mpi_request[std::max(N_GPUs, max_pair_num)];        // for cuda-aware MPI

    for (int cp = 0; cp < root.size(); ++ cp) {
        if (! check_pattern(root[cp], N_GPUs)) {
            continue;
        }
        if (rank == 0) {
            // Json::StyledWriter sw;
            Json::FastWriter sw;
            std::cout << sw.write(root[cp]);
            fflush(stdout);
        }
        for (int i = SIZEIDX_START; i < SIZEIDX_END; ++ i) {
            LL SIZE = SIZES[i];
            int** send_buf = new int*[N_GPUs];
            int** recv_buf = new int*[N_GPUs];
            // if (BACKEND.compare("hipMemcpy") == 0) {
            //     for (int gpuid = 0; gpuid < N_GPUs; ++ gpuid) {
            //         CUDA_CHECK(hipSetDevice(gpuid));
            //         CUDA_CHECK(hipMalloc(&send_buf[gpuid], SIZE * sizeof(int)));
            //         CUDA_CHECK(hipMalloc(&recv_buf[gpuid], SIZE * sizeof(int)));
            //     }
            // }
            if (BACKEND.compare("NCCL") == 0 || BACKEND.compare("MPI") == 0) {
                if (rank == 0) {
                    for (int gpuid = 0; gpuid < N_GPUs; ++ gpuid) {
                        CUDA_CHECK(hipSetDevice(gpuid));
                        CUDA_CHECK(hipMalloc(&send_buf[gpuid], SIZE * sizeof(int)));
                        CUDA_CHECK(hipMalloc(&recv_buf[gpuid], SIZE * sizeof(int)));
                    }
                } else {
                    CUDA_CHECK(hipSetDevice(rank));
                    CUDA_CHECK(hipMalloc(&send_buf[rank], SIZE * sizeof(int)));
                    CUDA_CHECK(hipMalloc(&recv_buf[rank], SIZE * sizeof(int)));
                }
            }
            
            // hipEvent_t start_a2a, stop_a2a;
            // float elapsedTime;
            // CUDA_CHECK(hipEventCreate(&start_a2a));
            // CUDA_CHECK(hipEventCreate(&stop_a2a));
            
            double max_BD = 0, min_BD = std::numeric_limits<double>::max();
            int MIXED_P2P_NUM = pow(3, root[cp].size());
            if (rank == 0) {
                printf("MIXED_P2P_NUM: %d\n", MIXED_P2P_NUM);
            }
            for (int p2p_id = 0; p2p_id < MIXED_P2P_NUM; ++ p2p_id) {
                bool has_0 = false;
                for (int k = 0; k < root[cp].size(); ++ k) {        // no NCCL
                    if (p2p_id / (int)pow(3, k) % 3 == 0) {
                        has_0 = true;
                        break;
                    }
                }
                if (has_0) {
                    continue;
                }

                // enableP2P
                if (rank == 0) {
                    for (int k = 0; k < root[cp].size(); ++ k) {
                        if (p2p_id / (int)pow(3, k) % 3 == 1) {
                            enableP2P(root[cp][k][0].asInt(), root[cp][k][1].asInt());
                        }
                    }
                }
                // WARMUP
                for (int _ = 0; _ < WARMUP; ++ _) {
                    mixed_p2p_comm(root[cp], p2p_id, send_buf, recv_buf, SIZE, streams, rank, pp->comm, mpi_request);
                    // CUDA_CHECK(hipGetLastError());
                    barrier(BACKEND, N_GPUs);
                }

                // CUDA_CHECK(hipDeviceSynchronize());
                // MPI_Barrier(MPI_COMM_WORLD);
                // devicesSyncAll(N_GPUs);
                barrier(BACKEND, N_GPUs);

                // CUDA_CHECK(hipEventRecord(start_a2a, stream));
                auto t0 = std::chrono::high_resolution_clock::now();

                for (int _ = 0; _ < TIMES; ++ _) {
                    mixed_p2p_comm(root[cp], p2p_id, send_buf, recv_buf, SIZE, streams, rank, pp->comm, mpi_request);
                    // CUDA_CHECK(hipDeviceSynchronize());    // light-barrier, [WHY]: 会有性能提升！！！ 减少 comm contention ?
                    // MPI_Barrier(MPI_COMM_WORLD);            // cpu-barrier, 没有意义
                    // devicesSyncAll(N_GPUs);                 // barrier(= light-barrier + cpu-barrier)
                    // barrier(BACKEND, N_GPUs);
                }
                barrier(BACKEND, N_GPUs);

                auto t1 = std::chrono::high_resolution_clock::now();        // CORRECT
                // CUDA_CHECK(hipEventElapsedTime(&elapsedTime, start_a2a, stop_a2a));    // ms
                // if (true) {
                if (rank == 0) {
                    // double t_d = (double)elapsedTime / 1000;    // s
                    double t_d = (double)(std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0).count()) / pow(1000, 2);  // s
                    double calc = root[cp].size() * (double)SIZE * sizeof(int) * TIMES;      // B
                    double avg_bd = calc / t_d;
                    for (int k = 0; k < N_GPUs; ++ k) {
                        printf("%d", p2p_id / (int)pow(3, k) % 3);
                    }
                    max_BD = std::max(max_BD, avg_bd);
                    min_BD = std::min(min_BD, avg_bd);
                    printf(": time %lf s, REAL_BD %lf GB/s, SIZE %lf KB, max_BD: %lf GB/s\n", \
                            t_d, avg_bd / pow(1024, 3) , (double)SIZE * sizeof(int) / pow(1024, 1), max_BD / pow(1024, 3));
                    fflush(stdout);
                }

                // disableP2P
                if (rank == 0) {
                    for (int k = 0; k < root[cp].size(); ++ k) {
                        if (p2p_id / (int)pow(3, k) % 3 == 1) {
                            disableP2P(root[cp][k][0].asInt(), root[cp][k][1].asInt());
                        }
                    }
                }
            }


            // if (BACKEND.compare("hipMemcpy") == 0) {
            //     for (int gpuid = 0; gpuid < N_GPUs; ++ gpuid) {
            //         CUDA_CHECK(hipFree(recv_buf[gpuid]));
            //         CUDA_CHECK(hipFree(send_buf[gpuid]));
            //     }
            // }
            if (BACKEND.compare("NCCL") == 0 || BACKEND.compare("MPI") == 0) {
                if (rank == 0) {
                    for (int gpuid = 0; gpuid < N_GPUs; ++ gpuid) {
                        CUDA_CHECK(hipFree(recv_buf[gpuid]));
                        CUDA_CHECK(hipFree(send_buf[gpuid]));
                    }
                } else {
                    CUDA_CHECK(hipFree(recv_buf[rank]));
                    CUDA_CHECK(hipFree(send_buf[rank]));
                }
            }
            
            delete[] recv_buf;
            delete[] send_buf;
        }
    }
    for (int i = 0; i < STREAM_NUM; ++ i) {
        CUDA_CHECK(hipStreamDestroy(streams[i]));
    }
    delete[] streams;
    // MPI_Finalize();
    return 0;
}