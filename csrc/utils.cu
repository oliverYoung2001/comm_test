#include "utils.h"
#include <set>
#include <unistd.h>

void barrier(std::string& BACKEND, int N_GPUs) {
    if (BACKEND.find("hipMemcpy") != std::string::npos) {
        for (int gpuid = 0; gpuid < N_GPUs; ++ gpuid) {
            CUDA_CHECK(hipSetDevice(gpuid));
            CUDA_CHECK(hipDeviceSynchronize());
        }
    }
    if (BACKEND.compare("NCCL") == 0 || BACKEND.compare("MPI") == 0) {
        CUDA_CHECK(hipDeviceSynchronize());
        MPI_Barrier(MPI_COMM_WORLD);
    }
}

void enableP2P(Json::Value& pairs) {
    // deduplicate
    std::set<std::pair<int, int> > s;
    for (int k = 0; k < pairs.size(); ++ k) {
        s.insert(std::make_pair(pairs[k][0].asInt(), pairs[k][1].asInt()));
    }
    // for (int k = 0; k < pairs.size(); ++ k) {
    for (auto it = s.begin(); it != s.end(); ++ it) {
        int src = it->first;
        int dst = it->second;
        CUDA_CHECK(hipSetDevice(src));
        int peer_access_available = 0;
        CUDA_CHECK(hipDeviceCanAccessPeer(&peer_access_available, src, dst));
        if (peer_access_available) {
            CUDA_CHECK(hipDeviceEnablePeerAccess(dst, 0));
        } else {
            printf("> GPU%d disabled direct access to GPU%d !!!\n", src, dst);
            fflush(stdout);
        }
    }
}

void disableP2P(Json::Value& pairs) {
    // deduplicate
    std::set<std::pair<int, int> > s;
    for (int k = 0; k < pairs.size(); ++ k) {
        s.insert(std::make_pair(pairs[k][0].asInt(), pairs[k][1].asInt()));
    }
    // for (int k = 0; k < pairs.size(); ++ k) {
    for (auto it = s.begin(); it != s.end(); ++ it) {
        int src = it->first;
        int dst = it->second;
        CUDA_CHECK(hipSetDevice(src));
        CUDA_CHECK(hipDeviceDisablePeerAccess(dst));
    }
}

void enableP2P(int ngpus) {
    for (int i = 0; i < ngpus; ++ i) {
        CUDA_CHECK(hipSetDevice(i));
        for (int j = 0; j < ngpus; ++ j) {
            if (i == j) {
                continue;
            }
            int peer_access_available = 0;
            CUDA_CHECK(hipDeviceCanAccessPeer(&peer_access_available, i, j));
            
            if (peer_access_available) {
                CUDA_CHECK(hipDeviceEnablePeerAccess(j, 0));
                // printf("> GPU%d enabled direct access to GPU%d\n", i, j);
                // fflush(stdout);
            } else {
                printf("> GPU%d disabled direct access to GPU%d !!!\n", i, j);
                fflush(stdout);
            }
        }
    }
}

void disableP2P(int ngpus) {
    for (int i = 0; i < ngpus; ++ i) {
        for (int j = 0; j < ngpus; ++ j) {
            if (i == j) {
                continue;
            }
            CUDA_CHECK(hipSetDevice(i));
            CUDA_CHECK(hipDeviceDisablePeerAccess(j));
        }
    }
}

void enableP2P(int i, int j) {
    if (i == j) {
        return;
    }
    CUDA_CHECK(hipSetDevice(i));
    int peer_access_available = 0;
    CUDA_CHECK(hipDeviceCanAccessPeer(&peer_access_available, i, j));
    if (peer_access_available) {
        CUDA_CHECK(hipDeviceEnablePeerAccess(j, 0));
    } else {
        printf("> GPU%d disabled direct access to GPU%d !!!\n", i, j);
        fflush(stdout);
    }  
}

void disableP2P(int i, int j) {
    CUDA_CHECK(hipSetDevice(i));
    CUDA_CHECK(hipDeviceDisablePeerAccess(j));
}

void check_UVA(int ngpus) {
    for (int gpuid = 0; gpuid < ngpus; ++ gpuid) {
        hipDeviceProp_t prop;
        CUDA_CHECK(hipGetDeviceProperties(&prop, gpuid));
        printf("GPU%d: %s unified addressing\n", gpuid, prop.unifiedAddressing ? "supports" : "does not support");
        fflush(stdout);
    }
}

// cudaMemcpy_comm: 不适用于多机
void cudaMemcpy_comm(PROC_PARAMS*& pp, Json::Value& pairs, int** send_buf, int** recv_buf, LL SIZE, \
               hipStream_t* streams, int rank, ncclComm_t comm, MPI_Request* mpi_request) {
#ifdef DIFF_BUF
    int* send_offset = new int[pp->N_GPUs];
    int* recv_offset = new int[pp->N_GPUs];
    memset(send_offset, 0, pp->N_GPUs * sizeof(int));
    memset(recv_offset, 0, pp->N_GPUs * sizeof(int));
    int src, dst;
    for (int k = 0; k < pairs.size(); ++ k) {
        src = pairs[k][0].asInt();
        dst = pairs[k][1].asInt();
        CUDA_CHECK(hipMemcpyPeerAsync(recv_buf[dst] + recv_offset[dst], dst, \
                                   send_buf[src] + send_offset[src], src, \
                                   SIZE * sizeof(int), streams[k]));                                // 两者性能相似
        send_offset[src] += SIZE;
        recv_offset[dst] += SIZE;
    }
    delete[] recv_offset;
    delete[] send_offset;
#else
    for (int k = 0; k < pairs.size(); ++ k) {
        // CUDA_CHECK(hipMemcpyAsync(recv_buf[pairs[k][1].asInt()], send_buf[pairs[k][0].asInt()], 
        //                             SIZE * sizeof(int), hipMemcpyDeviceToDevice, streams[k]));
        CUDA_CHECK(hipMemcpyPeerAsync(recv_buf[pairs[k][1].asInt()], pairs[k][1].asInt(), \
                                   send_buf[pairs[k][0].asInt()], pairs[k][0].asInt(), \
                                   SIZE * sizeof(int), streams[k]));                                // 两者性能相似
    }
#endif
}

// NCCL_comm
void NCCL_comm(PROC_PARAMS*& pp, Json::Value& pairs, int** send_buf, int** recv_buf, LL SIZE, \
               hipStream_t* streams, int rank, ncclComm_t comm, MPI_Request* mpi_request) {
    // printf("rank: %d", rank);
    // Json::FastWriter sw;
    // std::cout << sw.write(pairs);
    int send_offset = 0;
    int recv_offset = 0;
    NCCL_CHECK(ncclGroupStart());
    for (int k = 0; k < pairs.size(); ++ k) {
        if (rank == pairs[k][0].asInt()) {
            NCCL_CHECK(ncclSend(send_buf[rank] + send_offset, SIZE, ncclInt32, pairs[k][1].asInt(), comm, streams[0]));
    #ifdef DIFF_BUF
            send_offset += SIZE;
    #endif
        }
        if (rank == pairs[k][1].asInt()) {
            NCCL_CHECK(ncclRecv(recv_buf[rank] + recv_offset, SIZE, ncclInt32, pairs[k][0].asInt(), comm, streams[0]));
    #ifdef DIFF_BUF
            recv_offset += SIZE;
    #endif
        }
    }
    NCCL_CHECK(ncclGroupEnd());
}

// MPI_comm
void MPI_comm(PROC_PARAMS*& pp, Json::Value& pairs, int** send_buf, int** recv_buf, LL SIZE, \
               hipStream_t* streams, int rank, ncclComm_t comm, MPI_Request* mpi_request) {
    int req_num = 0;
    int send_offset = 0;
    int recv_offset = 0;
    for (int k = 0; k < pairs.size(); ++ k) {
        if (rank == pairs[k][0].asInt()) {
            MPI_Isend(send_buf[rank] + send_offset, SIZE, MPI_INT, pairs[k][1].asInt(), 0/*tag*/, MPI_COMM_WORLD, mpi_request + (req_num ++));
    #ifdef DIFF_BUF
            send_offset += SIZE;
    #endif
        }
        if (rank == pairs[k][1].asInt()) {
            MPI_Irecv(recv_buf[rank] + recv_offset, SIZE, MPI_INT, pairs[k][0].asInt(), 0/*tag*/, MPI_COMM_WORLD, mpi_request + (req_num ++));
    #ifdef DIFF_BUF
            recv_offset += SIZE;
    #endif
        }
    }
    MPI_Waitall(req_num, mpi_request , nullptr);

}

int parse_env(std::string key, std::string& value) {
    char* value0 = getenv(key.c_str());
    if (value0 == nullptr || strlen(value0) == 0) {
        return - 1;
    }
    value = std::string(value0);
    return 0;
}

int parse_env2int(std::string key, int& value) {
    std::string value_s;
    if (parse_env(key, value_s) != 0) {
        return - 1;
    }
    value = atoi(value_s.c_str());
    return 0;
}

void get_proc_params(PROC_PARAMS* pp) {
    // parse_env();
    parse_env("HOST", pp->host);
    if (parse_env2int("SLURM_PROCID", pp->rank) >= 0) { //  Use Slurm
        parse_env2int("SLURM_LOCALID", pp->local_rank);
        parse_env2int("SLURM_NTASKS", pp->comm_size);
        parse_env("SLURM_STEP_NODELIST", pp->ip);
        // hostname = socket.gethostname()
        // hostip = socket.gethostbyname(hostname)
        parse_env("SLURM_CLUSTER_NAME", pp->clustername);
        parse_env2int("SLURM_NODEID", pp->nodeid);
        parse_env("SLURMD_NODENAME", pp->nodename);
        parse_env2int("SLURM_TASKS_PER_NODE", pp->tasks_per_node);
    } else {    // Use Mpirun
        parse_env2int("OMPI_COMM_WORLD_RANK", pp->rank);
        parse_env2int("OMPI_COMM_WORLD_LOCAL_RANK", pp->local_rank);
        parse_env2int("OMPI_COMM_WORLD_SIZE", pp->comm_size);
        parse_env("OMPI_COMM_WORLD_HOSTNAME", pp->ip);  // None
        parse_env("OMPI_COMM_WORLD_CLUSTER_NAME", pp->clustername); // None
        parse_env2int("OMPI_COMM_WORLD_NODEID", pp->nodeid);    // None
        parse_env("OMPI_COMM_WORLD_NODENAME", pp->nodename);    // None
        parse_env2int("OMPI_COMM_WORLD_LOCAL_SIZE", pp->tasks_per_node);

    }
    pp->nodes = pp->comm_size / pp->tasks_per_node;     // default = 1
}

void setup_env(PROC_PARAMS*& pp, int argc, char** argv) {
    assert(argc >= 3);

    //Get number of gpus in the node
    int N_GPUs = std::stoi(argv[1]);
    std::string BACKEND = argv[2];
    
    pp = new PROC_PARAMS(N_GPUs);
    pp->BACKEND = BACKEND;

    // Init MPI
    // int comm_size, rank;
    if (pp->BACKEND.compare("NCCL") == 0 || pp->BACKEND.compare("MPI") == 0) {
        MPI_Init(&argc, &argv);
        // MPI_Init(NULL, NULL);
        MPI_Comm_size(MPI_COMM_WORLD, &pp->comm_size);
        MPI_Comm_rank(MPI_COMM_WORLD, &pp->rank);
        assert(pp->N_GPUs == pp->comm_size);
    }

    // pp->local_rank = pp->rank;              // default
    // pp->tasks_per_node = pp->comm_size;     // default

    get_proc_params(pp);
    // printf("rank: %d, local_rank: %d, comm_size: %d, tasks_per_node: %d\n", pp->rank, pp->local_rank, pp->comm_size, pp->tasks_per_node);
    if (pp->BACKEND.compare("NCCL") == 0 || pp->BACKEND.compare("MPI") == 0) {
        CUDA_CHECK(hipSetDevice(pp->local_rank));      // 至关重要！！！
    }

    // Init NCCL
    if (pp->BACKEND.compare("NCCL") == 0) {
        ncclUniqueId id;
        if (pp->rank == 0) ncclGetUniqueId(&id);
        MPI_Bcast(&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);
        ncclCommInitRank(&pp->comm, pp->comm_size, id, pp->rank);
    }
    if (pp->BACKEND.find("hipMemcpy") != std::string::npos) {
        pp->comm_size = 0;
        pp->rank = 0;
    }

    int GPU_VISIBLE;
    CUDA_CHECK(hipGetDeviceCount(&GPU_VISIBLE));
    assert(pp->tasks_per_node <= GPU_VISIBLE);
    assert(pp->N_GPUs <= GPU_VISIBLE * pp->nodes);

    // if (pp->rank == 0) {
    //     printf("BACKEND: %s\n", pp->BACKEND.c_str());
    //     fflush(stdout);
    // }

    parse_env("RECORD_P2P", pp->RECORD_P2P);

    if (pp->BACKEND.find("-P") != std::string::npos) {      // found "-P", ENABLE
        pp->ENABLE_GPU_P2P = true;
    }
}
