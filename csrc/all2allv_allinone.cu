#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include <assert.h>
#include <hipblas.h>
#include <cstdint>
#include <hiprand.h>
#include <mpi.h>
#include <string>
#include <vector>
#include <time.h>
#include <sys/time.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include "comm.h"
#include <assert.h>
#include <cmath>
#include <chrono>
#include <unistd.h>
#include <iostream>
#include "json/json.h"
#include <fstream>
#include "assert.h" 
#include "utils.h"

// #include <format>    // need c++20
typedef long long LL;

PROC_PARAMS* pp;

// #define CHECK_RESULT
// #define PRINT_JSON
int TIMES = 2000;
int WARMUP = 10;
const int MAGIC_FACTOR = pow(2, 5) * pow(3, 3) * pow(5, 2) * 7;     // 151200, for tests on different number of GPUs
// 62792 B

const int SIZEIDX_START = 4;
const int SIZEIDX_END = 20;

// const int SIZES_LEN = 8;
// const LL SIZES[SIZES_LEN] = {   // int = 4B
//     1LL * 256,                  // 1KB      // 打不满带宽
//     1LL * 1024 * 1,             // 4KB      // 打不满带宽
//     1LL * 1024 * 2,             // 8KB     // 会高一些!!! (仅在某些情况下)
//     1LL * 1024 * 4,             // 16KB     // 会高一些!!!  （最好）
//     // 1LL * 1024 * 8,             // 32KB     // 会高一些!!!  （最好）
//     1LL * 1024 * 16,            // 64KB     // 会高一些!!!  （最好）
//     // 1LL * 1024 * 64,            // 256KB    // 趋于稳定
//     1LL * 1024 * 256,           // 1MB
//     // 1LL * 1024 * 1024 * 1,      // 4MB      // 打不满带宽
//     1LL * 1024 * 1024 * 32,     // 128MB
//     // 1LL * 1024 * 1024 * 64,     // 256MB
//     1LL * 1024 * 1024 * 128,    // 512MB
//     // 1LL * 1024 * 1024 * 256,    // 1GB
//     // 1LL * 1024 * 1024 * 512,    // 2GB
//     // 1LL * 1024 * 1024 * 1024,   // 4GB      // 用hipMemcpy，竟然有性能下降！！！
//     // 1LL * 1024 * 1024 * 2048,   // 8GB
//     // 1LL * 1024 * 1024 * 4096,   // 16GB
//     // 1LL * 1024 * 1024 * 8192,   // OOM
// };

const int SIZES_LEN = 26;
const LL SIZES[SIZES_LEN] = {   // int = 4B
    1LL * 256,                  // 1KB
    1LL * 512,                  // 2KB
    1LL * 1024 * 1,             // 4KB
    1LL * 1024 * 2,             // 8KB
    1LL * 1024 * 4,             // 16KB
    1LL * 1024 * 8,             // 32KB
    1LL * 1024 * 16,            // 64KB
    1LL * 1024 * 32,            // 128KB
    1LL * 1024 * 64,            // 256KB
    1LL * 1024 * 128,           // 512KB
    1LL * 1024 * 256,           // 1MB
    1LL * 1024 * 512,           // 2MB
    1LL * 1024 * 1024 * 1,      // 4MB
    1LL * 1024 * 1024 * 2,      // 8MB
    1LL * 1024 * 1024 * 4,      // 16MB
    1LL * 1024 * 1024 * 8,      // 32MB
    1LL * 1024 * 1024 * 16,     // 64MB
    1LL * 1024 * 1024 * 32,     // 128MB
    1LL * 1024 * 1024 * 64,     // 256MB
    1LL * 1024 * 1024 * 128,    // 512MB
    1LL * 1024 * 1024 * 256,    // 1GB
    1LL * 1024 * 1024 * 512,    // 2GB
    1LL * 1024 * 1024 * 1024,   // 4GB
    1LL * 1024 * 1024 * 2048,   // 8GB
    1LL * 1024 * 1024 * 4096,   // 16GB
    1LL * 1024 * 1024 * 8192,   // OOM
};

// const int SIZES_LEN = 18;
// const LL SIZES[SIZES_LEN] = {           // int = 4B
//     (LL)MAGIC_FACTOR * 1,               // 590.6KB
//     (LL)MAGIC_FACTOR * 2,               
//     (LL)MAGIC_FACTOR * 4,           
//     (LL)MAGIC_FACTOR * 8,            
//     (LL)MAGIC_FACTOR * 16,             
//     (LL)MAGIC_FACTOR * 32,
//     (LL)MAGIC_FACTOR * 64,
//     (LL)MAGIC_FACTOR * 128,
//     (LL)MAGIC_FACTOR * 256,
//     (LL)MAGIC_FACTOR * 512,
//     (LL)MAGIC_FACTOR * 1024,
//     (LL)MAGIC_FACTOR * 1024 * 2,
//     (LL)MAGIC_FACTOR * 1024 * 4,
//     (LL)MAGIC_FACTOR * 1024 * 8,
//     (LL)MAGIC_FACTOR * 1024 * 16,
//     (LL)MAGIC_FACTOR * 1024 * 32,       // 18.46GB
//     (LL)MAGIC_FACTOR * 1024 * 64,       // 36.91GB
//     (LL)MAGIC_FACTOR * 1024 * 128,      // 73.82GB
// };

bool check_pattern(Json::Value pattern, int N_GPUs) {
    for (int k = 0; k < pattern.size(); ++ k) {
        if (std::max(pattern[k][0].asInt(), pattern[k][1].asInt()) >= N_GPUs) {
            return false;
        }
    }
    return true;
}

// void devicesSyncAll(int N_GPUs) {
//     for (int gpuid = 0; gpuid < N_GPUs; ++ gpuid) {
//         CUDA_CHECK(hipSetDevice(gpuid));
//         CUDA_CHECK(hipDeviceSynchronize());
//     }
// }

void barrier(std::string& BACKEND, int N_GPUs) {
    if (pp->BACKEND.find("hipMemcpy") != std::string::npos) {
        for (int gpuid = 0; gpuid < N_GPUs; ++ gpuid) {
            CUDA_CHECK(hipSetDevice(gpuid));
            CUDA_CHECK(hipDeviceSynchronize());
        }
    }
    if (BACKEND.compare("NCCL") == 0 || BACKEND.compare("MPI") == 0) {
        CUDA_CHECK(hipDeviceSynchronize());
        MPI_Barrier(MPI_COMM_WORLD);
    }
}

void MPI_v(int** sendcounts, int** recvcounts, int** sdispls, int** rdispls, int** send_buf, int** recv_buf, \
              hipStream_t* streams, int rank, ncclComm_t comm, MPI_Request* mpi_request) {
    MPI_Alltoallv(send_buf[rank], sendcounts[rank], sdispls[rank], MPI_BYTE, \
                  recv_buf[rank], recvcounts[rank], rdispls[rank], MPI_BYTE, MPI_COMM_WORLD);
}

void NCCL_v(int** sendcounts, int** recvcounts, int** sdispls, int** rdispls, int** send_buf, int** recv_buf, \
              hipStream_t* streams, int rank, ncclComm_t comm, MPI_Request* mpi_request) {
    NCCL_CHECK(ncclGroupStart());
    for (int k = 0; k < pp->N_GPUs; ++ k) {
        NCCL_CHECK(ncclSend((char*)send_buf[rank] + sdispls[rank][k], sendcounts[rank][k], ncclChar, k, comm, streams[0]));
        NCCL_CHECK(ncclRecv((char*)recv_buf[rank] + rdispls[rank][k], recvcounts[rank][k], ncclChar, k, comm, streams[0]));
    }
    NCCL_CHECK(ncclGroupEnd());
}

int main(int argc, char** argv) {
    if (argc < 4) {
        printf("Need at least 4 args: \"<command> <gpus> <backend> <cp_file>\"\n");
        return - 1;
    }
    setup_env(pp, argc, argv);
    std::string cp_file = argv[3];

    void (*XXX_v)(int** sendcounts, int** recvcounts, int** sdispls, int** rdispls, int** send_buf, int** recv_buf, \
              hipStream_t* streams, int rank, ncclComm_t comm, MPI_Request* mpi_request);
    if (pp->BACKEND.compare("NCCL") == 0) {
        XXX_v = NCCL_v;
    } else if (pp->BACKEND.compare("MPI") == 0) {
        XXX_v = MPI_v;
    } else if (pp->BACKEND.find("hipMemcpy") != std::string::npos) {
        // XXX_comm = cudaMemcpy_comm;
    } else {
        printf("Error BACKEND !!!");
        exit(- 1);
    }

    Json::Reader reader;
	Json::Value root;
    // std::string cp_file = "csrc/configs/conflict_patterns.json";
    std::ifstream in(cp_file.c_str(), std::ios::binary);
    if (! in.is_open()) {
		std::cout << "Error OPENING FILE\n";
		return - 1;
	}
    if (! reader.parse(in, root)) {
        std::cout << "Error READING FILE\n";
		return - 2;
    }

    // Init cudaStream
    int max_pair_num = 0;
    for (int cp = 0; cp < root.size(); ++ cp) {
        max_pair_num = std::max(max_pair_num, (int)root[cp].size());
    }
    pp->init_cudaStream(std::max(pp->N_GPUs, max_pair_num));

    // Init MPI_Request
    pp->init_MPI_Request(std::max(pp->N_GPUs, max_pair_num));


    // check_UVA(N_GPUs);        // 我理解，统一内存编址是为了方便，而不是性能

    int** sendcounts = new int*[pp->N_GPUs];
    int** recvcounts = new int*[pp->N_GPUs];
    int** sdispls = new int*[pp->N_GPUs];
    int** rdispls = new int*[pp->N_GPUs];
    for (int i = 0; i < pp->N_GPUs; ++ i) {
        sendcounts[i] = new int[pp->N_GPUs];
        recvcounts[i] = new int[pp->N_GPUs];
        sdispls[i] = new int[pp->N_GPUs + 1];
        rdispls[i] = new int[pp->N_GPUs + 1];
    }

    for (int cp = 0; cp < root.size(); ++ cp) {
        if (pp->rank == 0) {
            // Json::StyledWriter sw;
            Json::FastWriter sw;
            std::cout << sw.write(root[cp]);
            fflush(stdout);
        }
        if (pp->BACKEND.find("hipMemcpy") != std::string::npos && pp->ENABLE_GPU_P2P) {
            printf("[ERROR]: No Implementation !!!");
            exit(- 1);
            // enableP2P(root[cp]);
        }

        Json::Value commv = root[cp];
        int commv_sum = 0;
        for (int i = 0; i < pp->N_GPUs; ++ i) {
            for (int j = 0; j < pp->N_GPUs; ++ j) {
                // commv_max = std::max(commv_max, commv[i][j].asInt());
                commv_sum += commv[i][j].asInt();
            }
        }
        if (pp->rank == 0) {
            printf("commv_sum: %d\n", commv_sum);
        }

        // for (int __ = SIZEIDX_START; __ < SIZEIDX_END; ++ __) {

            for (int i = 0; i < pp->N_GPUs; ++ i) {
                sdispls[i][0] = 0;
                rdispls[i][0] = 0;
                for (int j = 0; j < pp->N_GPUs; ++ j) {
                    sendcounts[i][j] = commv[i][j].asInt();
                    recvcounts[i][j] = commv[j][i].asInt();
                    sdispls[i][j + 1] = sdispls[i][j] + sendcounts[i][j];
                    rdispls[i][j + 1] = rdispls[i][j] + recvcounts[i][j];
                }
                // if (pp->rank == 0) {
                //     printf("%d %d\n", sdispls[i][pp->N_GPUs], rdispls[i][pp->N_GPUs]);
                // }
            }

            int** send_buf = new int*[pp->N_GPUs];
            int** recv_buf = new int*[pp->N_GPUs];
            if (pp->BACKEND.find("hipMemcpy") != std::string::npos) {
                printf("[ERROR]: No Implementation !!!");
                exit(- 1);
                // for (int gpuid = 0; gpuid < pp->N_GPUs; ++ gpuid) {
                //     CUDA_CHECK(hipSetDevice(gpuid));
                //     CUDA_CHECK(hipMalloc(&send_buf[gpuid], SIZE * sizeof(int)));
                //     CUDA_CHECK(hipMalloc(&recv_buf[gpuid], SIZE * sizeof(int)));
                // }
            }
            if (pp->BACKEND.compare("NCCL") == 0 || pp->BACKEND.compare("MPI") == 0) {
                CUDA_CHECK(hipMalloc(&send_buf[pp->rank], sdispls[pp->rank][pp->N_GPUs]));
                CUDA_CHECK(hipMalloc(&recv_buf[pp->rank], rdispls[pp->rank][pp->N_GPUs]));
            }

            // WARMUP
            for (int _ = 0; _ < WARMUP; ++ _) {
                XXX_v(sendcounts, recvcounts, sdispls, rdispls, \
                        send_buf, recv_buf, pp->streams, pp->rank, pp->comm, pp->mpi_requests);
                barrier(pp->BACKEND, pp->N_GPUs);
            }

            barrier(pp->BACKEND, pp->N_GPUs);

            auto t0 = std::chrono::high_resolution_clock::now();

            for (int _ = 0; _ < TIMES; ++ _) {
                XXX_v(sendcounts, recvcounts, sdispls, rdispls, \
                        send_buf, recv_buf, pp->streams, pp->rank, pp->comm, pp->mpi_requests);
                barrier(pp->BACKEND, pp->N_GPUs);
            }
            barrier(pp->BACKEND, pp->N_GPUs);

            auto t1 = std::chrono::high_resolution_clock::now();        // CORRECT
            // CUDA_CHECK(hipEventElapsedTime(&elapsedTime, start_a2a, stop_a2a));    // ms
            // if (true) {
            if (pp->rank == 0) {
                // double t_d = (double)elapsedTime / 1000;    // s
                double t_d = (double)(std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0).count()) / pow(1000, 2);  // s
                double calc = 1 * (double)commv_sum * 1 * TIMES;      // B
                double avg_bd = calc / t_d / pow(1024, 3);
                printf("time %lf s, REAL_BD %lf GB/s, SIZE %lf KB, comm_vol %lf KB\n", \
                        t_d, avg_bd, (double)commv_sum * 1 / pow(1024, 1), calc / pow(1024, 1));
                fflush(stdout);
            }
            
            if (pp->BACKEND.find("hipMemcpy") != std::string::npos) {
                for (int gpuid = 0; gpuid < pp->N_GPUs; ++ gpuid) {
                    CUDA_CHECK(hipFree(recv_buf[gpuid]));
                    CUDA_CHECK(hipFree(send_buf[gpuid]));
                }
            }
            if (pp->BACKEND.compare("NCCL") == 0 || pp->BACKEND.compare("MPI") == 0) {
                CUDA_CHECK(hipFree(recv_buf[pp->rank]));
                CUDA_CHECK(hipFree(send_buf[pp->rank]));
            }
            
            delete[] recv_buf;
            delete[] send_buf;
        }
        if (pp->BACKEND.find("hipMemcpy") != std::string::npos && pp->ENABLE_GPU_P2P) {
            printf("[ERROR]: No Implementation !!!");
            exit(- 1);
            // disableP2P(root[cp]);
        }
    // }

    for (int i = 0; i < pp->N_GPUs; ++ i) {
        delete[] sendcounts[i];
        delete[] recvcounts[i];
        delete[] sdispls[i];
        delete[] rdispls[i];
    }
    delete[] sendcounts;
    delete[] recvcounts;
    delete[] sdispls;
    delete[] rdispls;
    
    delete pp;
    // MPI_Finalize();
    return 0;
}